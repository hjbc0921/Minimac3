#include "hip/hip_runtime.h"

#include "MarkovModel.h"
#include <hip/hip_runtime.h>

#define ThreadsPerBlock 256

__global__ void from_to(float *from, float *to, int *noReduced, int *unique, double *sum, double *com) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < *noReduced) {
        to[index] = from[index]*(*com)+(unique[index]*(*sum));
    }
}


void MarkovModel::initializeMatrices(HaplotypeSet & rHap,HaplotypeSet &tHap)
{
    if(LowMemory)
    {

        junctionLeftFoldProb.clear();
        junctionLeftFoldProb.resize(rHap.NoBlocks);
        for(int i=0;i<rHap.NoBlocks;i++)
        {
            junctionLeftFoldProb[i].resize(rHap.maxRepSize);
        }

        ThisBlockLeftProb.clear();
        ThisBlockLeftProb.resize(rHap.maxBlockSize);
        for(int i=0;i<rHap.maxBlockSize;i++)
        {
            ThisBlockLeftProb[i].resize(rHap.maxRepSize);
        }

    }
    else
    {
        leftProb.resize(rHap.NoBlocks);
        for(int i=0;i<rHap.NoBlocks;i++)
        {
            ReducedHaplotypeInfo &TempBlock=rHap.ReducedStructureInfo[i];
            vector<vector<float> > &TempLeft=leftProb[i];
            TempLeft.resize(TempBlock.BlockSize);
            for(int j=0;j<TempBlock.BlockSize;j++)
            {
                TempLeft[j].resize(TempBlock.RepSize);
            }
        }
    }


    CurrentLeftNoRecoProb.clear();
    CurrentLeftNoRecoProb.resize(rHap.maxRepSize);

    ThisBlockLeftNoRecoProb.clear();
    ThisBlockLeftNoRecoProb.resize(rHap.maxBlockSize);
    for(int i=0;i<rHap.maxBlockSize;i++)
    {
        ThisBlockLeftNoRecoProb[i].resize(rHap.maxRepSize);
    }

    junctionLeftProb.clear();
    junctionLeftProb.resize(rHap.NoBlocks+1);
    PrevjunctionRightProb.clear();
    PrevjunctionRightProb.resize(rHap.numHaplotypes);

    for(int i=0;i<=rHap.NoBlocks;i++)
    {
        junctionLeftProb[i].resize(rHap.numHaplotypes);
    }

    probHap.resize(rHap.maxRepSize);
    Constants.resize(rHap.maxRepSize);
    tempRightProb.reserve(rHap.maxRepSize);



}

void MarkovModel::ReinitializeMatrices()
{

    NoPrecisionJumps=0;
    fill(junctionLeftProb[0].begin(), junctionLeftProb[0].end(), 1.0);
    fill(PrevjunctionRightProb.begin(), PrevjunctionRightProb.end(), 1.0);

}



double MarkovModel::CountErrors(vector<float> &probHap,
                                int position, bool observed, double e,double freq,
                                ReducedHaplotypeInfo &Info)
{

    double match = 0;
    double mismatch = 0;
    double background = 0;

    for (int i = 0; i < noReducedStatesCurrent; i++)
    {

        if(Info.returnHapAtPosition(i,position)==observed)
            match += probHap[i];
        else
            mismatch += probHap[i];
    }


    background = (match + mismatch) * backgroundError;
    mismatch = (match + mismatch) * e *freq;
    match *= 1.0 - e;

    return mismatch / (mismatch + match + background);
}


double MarkovModel::CountRecombinants(vector<float> &from, vector<float> &to,
                                      vector<float> &probHap,double r,bool PrecisionMultiply)
{
    if (r == 0)
      return 0.0;

    double fromSum = 0.0,toSum=0.0,totalSum=0.0;

    for (int i = 0; i < noReducedStatesCurrent; i++)
    {
        fromSum += from[i];
        toSum += to[i];
        totalSum+=probHap[i];
    }

    double rsum = fromSum*r*toSum/(double)refCount;

    if(PrecisionMultiply)
        return (1e15*rsum / totalSum);
    else
        return (rsum / totalSum);
}



void MarkovModel::ReCreateLeftNoRecoProb(HaplotypeSet &tHap, int &hapID,
                           int group, ReducedHaplotypeInfo &Info,
                           vector<double> &alleleFreq)
{

    int &Start=Info.startIndex;
    int &End=Info.endIndex;
    noReducedStatesCurrent=Info.RepSize;
    ThisBlockLeftNoRecoProb[0]=leftProb[group][0];

    for (int markerPos=Start+1; markerPos<=End; markerPos++)
    {
        vector<float> &NextnoRecomProb = ThisBlockLeftNoRecoProb[markerPos-Start];
        double complement = 1. - Recom[markerPos-1];
        NextnoRecomProb=ThisBlockLeftNoRecoProb[markerPos-Start-1];
        double freq=tHap.getScaffoldedHaplotype(hapID,markerPos)? alleleFreq[markerPos] : 1-alleleFreq[markerPos];
        double e=Error[markerPos];
        bool observed=tHap.getScaffoldedHaplotype(hapID,markerPos);
        double prandom = e*freq+backgroundError;
        double pmatch = (1.0 - e)+e*freq+backgroundError;

        for (int i = 0; i <noReducedStatesCurrent; i++)
        {
            NextnoRecomProb[i]*=(complement);
        }

        if (PrecisionJump[markerPos])
        {
            for (int i = 0; i <noReducedStatesCurrent; i++)
            {
                NextnoRecomProb[i]*=(1e15);
            }
        }

        if (!missing[markerPos] && !tHap.getMissingScaffoldedHaplotype(hapID,markerPos))
        {
            for (int i = 0; i<noReducedStatesCurrent; i++)
                {
                    NextnoRecomProb[i]*=(Info.returnHapAtPosition(i,markerPos)==observed)?pmatch:prandom;
                }
        }

    }
}





void MarkovModel::ReCreateBothLeftProb(HaplotypeSet &tHap, int &hapID,
                           int group, ReducedHaplotypeInfo &Info,
                           vector<double> &alleleFreq)
{

    vector<vector<float> > &Leftprob = ThisBlockLeftProb;
    Leftprob[0]=junctionLeftFoldProb[group];
    ThisBlockLeftNoRecoProb[0]=Leftprob[0];

    int &Start=Info.startIndex;
    int &End=Info.endIndex;

    noReducedStatesCurrent=Info.RepSize;
    for (int markerPos=Start+1; markerPos<=End; markerPos++)
    {
        ThisBlockLeftNoRecoProb[markerPos-Start]=ThisBlockLeftNoRecoProb[markerPos-Start-1];
        time_t t0 = time(NULL);
        Transpose(Leftprob[markerPos-Start-1],
                  Leftprob[markerPos-Start],ThisBlockLeftNoRecoProb[markerPos-Start],
                  Recom[markerPos-1],Info.uniqueCardinality);
        time_t t1 = time(NULL);
        transposeTime += (t1-t0);
        if (!missing[markerPos] && !tHap.getMissingScaffoldedHaplotype(hapID,markerPos))
        {

           if(!tHap.getMissingScaffoldedHaplotype(hapID,markerPos))
                Condition(markerPos,Leftprob[markerPos-Start],
                     ThisBlockLeftNoRecoProb[markerPos-Start],
                     tHap.getScaffoldedHaplotype(hapID,markerPos),
                     Error[markerPos],
                     tHap.getScaffoldedHaplotype(hapID,markerPos)?
                          alleleFreq[markerPos] : 1-alleleFreq[markerPos],Info);

        }

    }
}




void MarkovModel::CountExpected(HaplotypeSet &tHap,int hapID,int group,
                                   vector<float> &PrevRightFoldedProb,
                                    vector<float> &CurrentRightProb, vector<float> &CurrentNoRecoRightProb,
                                    ReducedHaplotypeInfo &Info,vector<double> &alleleFreq)
{


    vector<float> &juncLeftprob = junctionLeftProb[group];
    vector<float> &juncRightProb = PrevjunctionRightProb;

    if(LowMemory)
        ReCreateBothLeftProb(tHap,hapID,group,Info,alleleFreq);
    else
        ReCreateLeftNoRecoProb(tHap,hapID,group,Info,alleleFreq);

    vector<vector<float> > &Leftprob = LowMemory? ThisBlockLeftProb: leftProb[group];
    vector<vector<float> > &leftNoRecomProb= ThisBlockLeftNoRecoProb;


    int &Start=Info.startIndex;
    int &End=Info.endIndex;

    CurrentRightProb=PrevRightFoldedProb;
    CurrentNoRecoRightProb=PrevRightFoldedProb;
    noReducedStatesCurrent=Info.RepSize;

    fill(Constants.begin(), Constants.end(), 0.0);
    for(int i=0;i<refCount;i++)
        Constants[Info.uniqueIndexMap[i]]+=(juncLeftprob[i]*juncRightProb[i]);


    CreatePosteriorProb(Leftprob[End-Start],CurrentRightProb,leftNoRecomProb[End-Start],
                        CurrentNoRecoRightProb,Leftprob[0],PrevRightFoldedProb,Constants,
                        probHap,Info);

    if(!missing[End] && !tHap.getMissingScaffoldedHaplotype(hapID,End))
         empError[End]+=CountErrors(probHap,End,tHap.getScaffoldedHaplotype(hapID,End),
                               Error[End],
                               tHap.getScaffoldedHaplotype(hapID,End)? alleleFreq[End] : 1-alleleFreq[End],Info);
    else
        empError[End]+=Error[End];

    for (int markerPos=End-1; markerPos>Start; markerPos--)
    {
        if (!missing[markerPos+1] && !tHap.getMissingScaffoldedHaplotype(hapID,markerPos+1))
        {

              Condition(markerPos+1,CurrentRightProb,CurrentNoRecoRightProb,
                      tHap.getScaffoldedHaplotype(hapID,markerPos+1),
                      Error[markerPos+1],
                       tHap.getScaffoldedHaplotype(hapID,markerPos+1)?
                          alleleFreq[markerPos+1] : 1-alleleFreq[markerPos+1],Info);
        }
        tempRightProb=CurrentRightProb;

        empRecom[markerPos]+=CountRecombinants(Leftprob[markerPos-Start],CurrentRightProb,probHap,
                                               Recom[markerPos]
                                               ,PrecisionJump[markerPos+1]);
        time_t t0 = time(NULL);
        Transpose(tempRightProb,CurrentRightProb,CurrentNoRecoRightProb,Recom[markerPos],Info.uniqueCardinality);
        time_t t1 = time(NULL);
        transposeTime += (t1-t0);
        CreatePosteriorProb(Leftprob[markerPos-Start],CurrentRightProb,
                            leftNoRecomProb[markerPos-Start],CurrentNoRecoRightProb,
                            Leftprob[0],PrevRightFoldedProb,Constants,probHap,Info);


        if(!missing[markerPos] && !tHap.getMissingScaffoldedHaplotype(hapID,markerPos))
            empError[markerPos]+=CountErrors(probHap,markerPos,
                                             tHap.getScaffoldedHaplotype(hapID,markerPos),
                               Error[markerPos],
                               tHap.getScaffoldedHaplotype(hapID,markerPos)?
                                              alleleFreq[markerPos] : 1-alleleFreq[markerPos],Info);
        else
            empError[markerPos]+=Error[markerPos];

    }

    if (!missing[Start+1] && !tHap.getMissingScaffoldedHaplotype(hapID,Start+1))
        {

              Condition(Start+1,CurrentRightProb,CurrentNoRecoRightProb,
                      tHap.getScaffoldedHaplotype(hapID,Start+1),
                      Error[Start+1],
                       tHap.getScaffoldedHaplotype(hapID,Start+1)?
                          alleleFreq[Start+1] : 1-alleleFreq[Start+1],Info);
        }

    tempRightProb=CurrentRightProb;
    empRecom[Start]+=CountRecombinants(Leftprob[0],CurrentRightProb,probHap,Recom[Start],PrecisionJump[Start+1]);
    time_t t0 = time(NULL);
    Transpose(tempRightProb,CurrentRightProb,CurrentNoRecoRightProb,Recom[Start],Info.uniqueCardinality);
    time_t t1 = time(NULL);
    transposeTime += (t1-t0);

    if(Start==0)
        {

            CreatePosteriorProb(Leftprob[0],CurrentRightProb,leftNoRecomProb[0],
                                CurrentNoRecoRightProb,Leftprob[0],PrevRightFoldedProb,
                                Constants,probHap,Info);

            if(!missing[Start] && !tHap.getMissingScaffoldedHaplotype(hapID,Start))
                 empError[Start]+=CountErrors(probHap,Start,tHap.getScaffoldedHaplotype(hapID,Start),
                                       Error[Start],
                                       tHap.getScaffoldedHaplotype(hapID,Start)? alleleFreq[Start] : 1-alleleFreq[Start]
                                       ,Info);
            else
                empError[Start]+=Error[Start];

        }
}



void MarkovModel::Impute(HaplotypeSet &tHap,int hapID,int group,
                                   vector<float> &PrevRightFoldedProb,
                                    vector<float> &CurrentRightProb, vector<float> &CurrentNoRecoRightProb,
                                    ReducedHaplotypeInfo &Info,vector<double> &alleleFreq)
{

    vector<float> &juncLeftprob = junctionLeftProb[group];
    vector<float> &juncRightProb = PrevjunctionRightProb;


    if(LowMemory)
        ReCreateBothLeftProb(tHap,hapID,group,Info,alleleFreq);
    else
        ReCreateLeftNoRecoProb(tHap,hapID,group,Info,alleleFreq);

    vector<vector<float> > &Leftprob = LowMemory? ThisBlockLeftProb: leftProb[group];
    vector<vector<float> > &leftNoRecomProb= ThisBlockLeftNoRecoProb;

    int &start=Info.startIndex;
    int &end=Info.endIndex;


    CurrentRightProb=PrevRightFoldedProb;
    CurrentNoRecoRightProb=PrevRightFoldedProb;


    fill(Constants.begin(), Constants.end(), 0.0);
    noReducedStatesCurrent=Info.RepSize;
    for(int i=0;i<refCount;i++)
            Constants[Info.uniqueIndexMap[i]]+=(juncLeftprob[i]*juncRightProb[i]);

    Impute(end,tHap.getScaffoldedHaplotype(hapID,end),tHap.getMissingScaffoldedHaplotype(hapID,end),Leftprob[end-start],
           CurrentRightProb,leftNoRecomProb[end-start],CurrentNoRecoRightProb,Leftprob[0],
               PrevRightFoldedProb,Constants,Info,alleleFreq);
    for (int markerPos=end-1; markerPos>start; markerPos--)
    {

         if (!missing[markerPos+1] && !tHap.getMissingScaffoldedHaplotype(hapID,markerPos+1))
        {

              Condition(markerPos+1,CurrentRightProb,CurrentNoRecoRightProb,
                      tHap.getScaffoldedHaplotype(hapID,markerPos+1),
                      Error[markerPos+1],
                       tHap.getScaffoldedHaplotype(hapID,markerPos+1)?
                          alleleFreq[markerPos+1] : 1-alleleFreq[markerPos+1],Info);
        }


        tempRightProb=CurrentRightProb;
         time_t t0 = time(NULL);
        Transpose(tempRightProb,CurrentRightProb,CurrentNoRecoRightProb,Recom[markerPos],
                            Info.uniqueCardinality);
        time_t t1 = time(NULL);
        transposeTime += (t1-t0);
        Impute(markerPos,tHap.getScaffoldedHaplotype(hapID,markerPos),tHap.getMissingScaffoldedHaplotype(hapID,markerPos),
               Leftprob[markerPos-start],CurrentRightProb,leftNoRecomProb[markerPos-start],
                   CurrentNoRecoRightProb,Leftprob[0],PrevRightFoldedProb,Constants,Info,alleleFreq);
    }

    if (!missing[start+1] && !tHap.getMissingScaffoldedHaplotype(hapID,start+1))
        {

              Condition(start+1,CurrentRightProb,CurrentNoRecoRightProb,
                      tHap.getScaffoldedHaplotype(hapID,start+1),
                      Error[start+1],
                       tHap.getScaffoldedHaplotype(hapID,start+1)?
                          alleleFreq[start+1] : 1-alleleFreq[start+1],Info);
        }

    tempRightProb=CurrentRightProb;
    time_t t0 = time(NULL);
    Transpose(tempRightProb,CurrentRightProb,CurrentNoRecoRightProb,Recom[start],Info.uniqueCardinality);
    time_t t1 = time(NULL);
    transposeTime += (t1-t0);

    if(start==0)
        Impute(start,tHap.getScaffoldedHaplotype(hapID,start),tHap.getMissingScaffoldedHaplotype(hapID,start),Leftprob[0],
               CurrentRightProb,leftNoRecomProb[0],
                       CurrentNoRecoRightProb,Leftprob[0],PrevRightFoldedProb,Constants,Info,alleleFreq);

}





void MarkovModel::CreatePosteriorProb(vector<float> &Leftprob,vector<float> &rightProb,
                         vector<float> &leftNoRecoProb,vector<float> &rightNoRecoProb,
                         vector<float> &leftEndProb,vector<float> &rightEndProb,
                         vector<float> &Constants,vector<float> &probHap,ReducedHaplotypeInfo &Info)
{

    double value=0.0;
    for(int i=0;i<noReducedStatesCurrent;i++)
    {

        value = Constants[i]*(leftNoRecoProb[i]*rightNoRecoProb[i]/(leftEndProb[i]*rightEndProb[i]))
            +(Leftprob[i]*rightProb[i]-leftNoRecoProb[i]*rightNoRecoProb[i])*(Info.InvuniqueCardinality[i]);

        probHap[i]=value;

    }
}




void MarkovModel::Impute(int position, bool observed, bool observedMiss,
                         vector<float> &Leftprob,vector<float> &rightProb,
                         vector<float> &leftNoRecoProb,vector<float> &rightNoRecoProb,
                         vector<float> &leftEndProb,vector<float> &rightEndProb,
                         vector<float> &Constants,ReducedHaplotypeInfo &Info,
                         vector<double> &alleleFreq)
{



    float Pref=0.0,Palt=0.0;

    float *value = (float *)alloca(noReducedStatesCurrent*sizeof(float));
    for(int i=0; i<noReducedStatesCurrent; i++)
    {
        // careful: order of operations is important to avoid overflows
        value[i] = Constants[i]*(leftNoRecoProb[i]*rightNoRecoProb[i]/(leftEndProb[i]*rightEndProb[i]))
            +(Leftprob[i]*rightProb[i]-leftNoRecoProb[i]*rightNoRecoProb[i])*(Info.InvuniqueCardinality[i]);
    }

    for (int i=0; i<noReducedStatesCurrent;)
    {
        bool hp = Info.returnHapAtPosition(i,position);

        float pp=0.0;

        pp= value[i] + (hp? Palt:Pref);

        i++;
        while ((i < noReducedStatesCurrent) && (hp == Info.returnHapAtPosition(i,position)))
        {
            pp += value[i];
            i++;
        }

        if(hp)
            Palt = pp ;
        else
            Pref = pp;
    }


    float ptotal =Pref+Palt;
    bool mle = false;

    if(Pref<Palt)
    {
        mle=true;
    }



    imputedDose[position] += imputedHap[position]= (Palt / ptotal);
    imputedAlleleNumber[position] = mle;

    if(!observedMiss)
    {

        double fmatch = 1.0 / (1. - Error[position] + Error[position] * ( major[position]? alleleFreq[position] : 1-alleleFreq[position]  ) + backgroundError );
        double fmismatch = 1.0 / (Error[position] * ( major[position]? alleleFreq[position] : 1-alleleFreq[position]  ) + backgroundError);

        if(observed)
        {
            Palt *= fmatch;
            Pref *= fmismatch;
        }
        else
        {
            Pref *= fmatch;
            Palt *= fmismatch;
        }

        ptotal =Pref+Palt;
        leaveOneOut[position] = Palt / ptotal;
    }


}




void MarkovModel::foldProbabilities(vector<float> &foldProb,int bridgeIndex,ReducedHaplotypeInfo &Info,int direction,int noReference) //0 - left; 1 - right
{
    vector<int> *TempuniqueIndexMap=&Info.uniqueIndexMap;
    fill(foldProb.begin(), foldProb.end(), 0.0);
    if(direction==0)
    {
        vector<float> *PrevjunctionLeftProb=&junctionLeftProb[bridgeIndex];
        for(int i=0;i<noReference;i++)
        {
            foldProb[(*TempuniqueIndexMap)[i]]+=(*PrevjunctionLeftProb)[i];
        }
    }
    else if(direction==1)
    {

        for(int i=0;i<noReference;i++)
        {
            foldProb[(*TempuniqueIndexMap)[i]]+=PrevjunctionRightProb[i];
        }
    }
}




void MarkovModel::unfoldProbabilities(int bridgeIndex,vector<float> &recomProb,
                                       vector<float> &noRecomProb,vector<float> &PrevFoldedProb,
                                     int direction,vector<ReducedHaplotypeInfo> &StructureInfo,
                                     int noReference)
{
    ReducedHaplotypeInfo &thisInfo = StructureInfo[bridgeIndex];
    int N = thisInfo.RepSize;

    float *adj_rec = (float *)alloca(N*sizeof(float));
    float *adj_norec = (float *)alloca(N*sizeof(float));

    for (int i=0; i<N; i++)
    {
        adj_rec[i] = recomProb[i] * thisInfo.InvuniqueCardinality[i];
    }
    for (int i=0; i<N; i++)
    {
        adj_norec[i] = noRecomProb[i] / PrevFoldedProb[i];
    }
    vector<float> &prev = direction ? PrevjunctionRightProb : junctionLeftProb[bridgeIndex];
    vector<float> &next = direction ? PrevjunctionRightProb : junctionLeftProb[bridgeIndex+1];

    if(direction)
    {
        for (int i=0; i<noReference; i++)
        {
            int m = thisInfo.uniqueIndexMap[i];
            prev[i]*=adj_norec[m];
            prev[i]+=adj_rec[m];
        }
    }
    else
    {
         for (int i=0; i<noReference; i++)
        {
            int m = thisInfo.uniqueIndexMap[i];
            next[i] = adj_rec[m] + adj_norec[m]*prev[i];
        }
    }

}



void MarkovModel::WalkLeft(HaplotypeSet &tHap, int &hapID,
                           int group, ReducedHaplotypeInfo &Info,
                           vector<double> &alleleFreq)
{
    vector<vector<float> > &Leftprob = LowMemory ? ThisBlockLeftProb : leftProb[group];
    if(LowMemory)
        junctionLeftFoldProb[group]=Leftprob[0];

    int &Start=Info.startIndex;
    int &End=Info.endIndex;

    noReducedStatesCurrent=Info.RepSize;
    for (int markerPos=Start+1; markerPos<=End; markerPos++)
    {
        cout << Leftprob[markerPos-Start][0] << endl;
        time_t t0 = time(NULL);
        PrecisionJump[markerPos]=Transpose(Leftprob[markerPos-Start-1],
                  Leftprob[markerPos-Start],CurrentLeftNoRecoProb,
                  Recom[markerPos-1],Info.uniqueCardinality);
        time_t t1 = time(NULL);
        transposeTime += (t1-t0);
        cout << Leftprob[markerPos-Start][0] << endl;
        if (!missing[markerPos] && !tHap.getMissingScaffoldedHaplotype(hapID,markerPos))
        {
           if(!tHap.getMissingScaffoldedHaplotype(hapID,markerPos))
                Condition(markerPos,Leftprob[markerPos-Start],
                     CurrentLeftNoRecoProb,
                     tHap.getScaffoldedHaplotype(hapID,markerPos),
                     Error[markerPos],
                     tHap.getScaffoldedHaplotype(hapID,markerPos)?
                          alleleFreq[markerPos] : 1-alleleFreq[markerPos],Info);
        }
    }
}





void MarkovModel::Condition(int markerPos,vector<float> &Prob,
                            vector<float> &noRecomProb, bool observed,double e,double freq,ReducedHaplotypeInfo &Info)
{

    double prandom = e*freq+backgroundError;
    double pmatch = (1.0 - e)+e*freq+backgroundError;

    for (int i = 0; i<noReducedStatesCurrent; i++)
    {

        bool allele=Info.returnHapAtPosition(i,markerPos);
        if(allele==observed)
        {
            Prob[i]*=pmatch;
            noRecomProb[i]*=pmatch;
        }
        else
        {
            Prob[i]*=prandom;
            noRecomProb[i]*=prandom;
        }
    }
}



bool MarkovModel::Transpose(vector<float> &from,
                            vector<float> &to, vector<float> &noRecomProb,
                            double reco,vector<int> &uniqueCardinality)
{
    bool tempPrecisionJumpFlag=false;
    if (reco == 0)
    {
        to=from;
        return false;
    }

    double sum = 0.0;
    for (int i = 0; i <noReducedStatesCurrent; i++)
    {
        sum += from[i];
        noRecomProb[i]*=(1.-reco);
    }

    sum*=(reco/(double)refCount);
    double complement = 1. - reco;

    // avoid underflows
    if (sum < 1e-10)
    {
        tempPrecisionJumpFlag=true;
        sum*= 1e15;
        complement *= 1e15;
        for(int i=0;i<noReducedStatesCurrent;i++)
            noRecomProb[i]*=1e15;
        NoPrecisionJumps++;
    }

    // GPU

    int size1 = noReducedStatesCurrent * sizeof(float);
    int size2 = noReducedStatesCurrent * sizeof(int);
    float *d_from, *d_to;
    int *d_noReduced, *d_unique;
    double *d_sum, *d_complement;

    float *t_from = from.data();
    float *t_to = to.data();
    int *t_unique = uniqueCardinality.data();

    hipMalloc((void **)&d_from, size1);
    hipMalloc((void **)&d_to, size1);
    hipMalloc((void **)&d_noReduced, sizeof(int));
    hipMalloc((void **)&d_unique, size2);
    hipMalloc((void **)&d_sum, sizeof(double));
    hipMalloc((void **)&d_complement, sizeof(double));

    hipMemcpy(d_from, t_from, size1, hipMemcpyHostToDevice);
    hipMemcpy(d_to, t_to, size1, hipMemcpyHostToDevice);
    hipMemcpy(d_noReduced, &noReducedStatesCurrent, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_unique, t_unique, size2, hipMemcpyHostToDevice);
    hipMemcpy(d_sum, &sum, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_complement, &complement, sizeof(double), hipMemcpyHostToDevice);

    from_to<<<(noReducedStatesCurrent + ThreadsPerBlock - 1)/ThreadsPerBlock, ThreadsPerBlock>>>(d_from, d_to, d_noReduced, d_unique, d_sum, d_complement);

    hipMemcpy(t_to, d_to, size1, hipMemcpyDeviceToHost);

    hipFree(d_from);
    hipFree(d_to);
    hipFree(d_noReduced);
    hipFree(d_unique);
    hipFree(d_sum);
    hipFree(d_complement);

    return tempPrecisionJumpFlag;


 }




void MarkovModel::CheckSize(HaplotypeSet & rHap,HaplotypeSet &tHap)
{

    cout.precision(4);
    double LeftBlockSize=0.0,LeftNoRecoSum = 0.0, TransSize=0.0, HapData=0.0;


    for(int i=0;i<rHap.NoBlocks;i++)
    {
        ReducedHaplotypeInfo &TempBlock=rHap.ReducedStructureInfo[i];


        HapData+=TempBlock.uniqueIndexMap.capacity()*sizeof(int);
        HapData+=TempBlock.uniqueCardinality.capacity()*sizeof(int);
        HapData+=TempBlock.InvuniqueCardinality.capacity()*sizeof(float);

        for(int j=0;j<TempBlock.RepSize;j++)
        {
            HapData+=TempBlock.uniqueHaps[j].capacity()*sizeof(bool);
        }
    }

    cout<<" HAP_DATA = "<<HapData/(1024*1024*1024)<<" ";




    if(LowMemory)
    {

        for(int i=0;i<rHap.NoBlocks;i++)
        {
            LeftBlockSize+=junctionLeftFoldProb[i].capacity()*sizeof(float);
        }


        cout<<" R_LEFT_BLOCK = "<<LeftBlockSize/(1024*1024*1024)<<" ";

        double temp=0.0;


        for(int i=0;i<rHap.maxBlockSize;i++)
        {
            temp+=ThisBlockLeftProb[i].capacity()*sizeof(float);
        }
        cout<<" R_LEFT_SAVE = "<<temp/(1024*1024*1024)<<" ";
        LeftBlockSize+=temp;
    }
    else
    {
        for(int i=0;i<rHap.NoBlocks;i++)
        {
            ReducedHaplotypeInfo &TempBlock=rHap.ReducedStructureInfo[i];
            vector<vector<float> > &TempLeft=leftProb[i];
            for(int j=0;j<TempBlock.BlockSize;j++)
            {
//                cout<<TempLeft[j].capacity()<<"\t"<<sizeof(float)<<endl;
                LeftBlockSize+=TempLeft[j].capacity()*sizeof(float);
            }
        }
    }



    cout<<" R_LEFT = "<<LeftBlockSize/(1024*1024*1024)<<" ";
    LeftNoRecoSum+=CurrentLeftNoRecoProb.capacity()*sizeof(float);
    for(int i=0;i<rHap.maxBlockSize;i++)
    {
        LeftNoRecoSum+=ThisBlockLeftNoRecoProb[i].capacity()*sizeof(float);
    }
    cout<<" NOR_LEFT = "<<LeftNoRecoSum/(1024*1024*1024)<<" ";
    cout<<" R_RIGHT = "<<tempRightProb.capacity()*sizeof(float)/(1024*1024*1024)<<" ";
    cout<<" NOR_RIGHT = "<<tempRightProb.capacity()*sizeof(float)/(1024*1024*1024)<<" ";

    LeftBlockSize+=LeftNoRecoSum;
    LeftBlockSize+=(2*tempRightProb.capacity()*sizeof(float));

    cout<<" BLOCK_SUM = "<<LeftBlockSize/(1024*1024*1024)<<" ";



    for(int i=0;i<=rHap.NoBlocks;i++)
    {
        TransSize+=junctionLeftProb[i].capacity()*sizeof(float);
    }

    cout<<" J_LEFT = "<<TransSize/(1024*1024*1024)<<" ";
    cout<<" J_RIGHT = "<<PrevjunctionRightProb.capacity()*sizeof(float)/(1024*1024*1024)<<" ";
    TransSize+=PrevjunctionRightProb.capacity()*sizeof(float);
    cout<<" J_SUM = "<<TransSize/(1024*1024*1024)<<" ";

    cout<<endl;


}




